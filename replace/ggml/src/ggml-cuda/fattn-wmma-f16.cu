#include "hip/hip_runtime.h"
// ggml/src/ggml-cuda/fattn-wmma-f16.cu (replacement)
#include "common.cuh"
#include "fattn-common.cuh"

extern "C" __global__ void flash_attn_ext_f16_stub() { /* noop */ }

void ggml_cuda_flash_attn_ext_wmma_f16(ggml_backend_cuda_context & ctx,
                                       ggml_tensor * dst) {
    GGML_UNUSED(ctx);
    GGML_UNUSED(dst);
}
